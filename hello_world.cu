#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "array_gen.h"

__global__ void hello_from_gpu() {
    printf("Hello from GPU!\n");
}

__global__ void splitArrayIntoThreads(int* input, int* output1, int* output2, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N / 2) {
        output1[idx] = input[2 * idx];       // First integer for the thread
        output2[idx] = input[2 * idx + 1];   // Second integer for the thread
    }
}

int main() {

    const int N = 10;
    int h_input[N];
    generate_random_array(h_input, N); //Example input array

    // Print input array
    for(int i = 0; i < N; i++) {
        printf("%d ", h_input[i]);
    }   
    // Allocate memory for output arrays
    int h_output1[N / 2];
    int h_output2[N / 2];

    int *d_input, *d_output1, *d_output2;

    // Allocate GPU memory
    hipMalloc((void**)&d_input, N * sizeof(int));
    hipMalloc((void**)&d_output1, (N / 2) * sizeof(int));
    hipMalloc((void**)&d_output2, (N / 2) * sizeof(int));

    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N / 2 + threadsPerBlock - 1) / threadsPerBlock;
    splitArrayIntoThreads<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output1, d_output2, N);

    // Copy output data back to host
    hipMemcpy(h_output1, d_output1, (N / 2) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_output2, d_output2, (N / 2) * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    printf("Thread outputs:\n");
    for (int i = 0; i < N / 2; i++) {
        printf("Thread %d: %d, %d\n", i, h_output1[i], h_output2[i]);
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output1);
    hipFree(d_output2);

    return 0;
}

//nvcc -o program program.cu