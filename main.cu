#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "array_gen.h"

// CUDA kernel for performing bitonic merge
__global__ void bitonic_merge(int *d_array, int low, int cnt, int dir) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Each thread processes two elements, comparing and swapping them
    int i = low + idx * 2;
    int j = i + cnt / 2;

    // Make sure indices are within bounds
    if (i < low + cnt - cnt / 2 && j < low + cnt) {
        if ((d_array[i] > d_array[j]) == dir) {
            // Swap elements if needed
            int temp = d_array[i];
            d_array[i] = d_array[j];
            d_array[j] = temp;
        }
    }
}

// CUDA kernel for performing bitonic sort
__global__ void bitonic_sort(int *d_array, int low, int cnt, int dir) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Each thread processes two elements
    int i = low + idx * 2;
    int j = i + cnt / 2;

    // Make sure indices are within bounds
    if (i < low + cnt - cnt / 2 && j < low + cnt) {
        if ((d_array[i] > d_array[j]) == dir) {
            // Swap elements if needed
            int temp = d_array[i];
            d_array[i] = d_array[j];
            d_array[j] = temp;
        }
    }
}

// Split function that works exactly as you mentioned
void split(int *arr, int low, int cnt, int dir) {
    for (int size = 2; size <= cnt; size = size * 2) {
        for (int i = low; i < low + cnt - size; i++) {
            int j = i + size / 2;
            if ((arr[i] > arr[j]) == dir) {
                int temp = arr[i];
                arr[i] = arr[j];
                arr[j] = temp;
            }
        }
    }
}

// Host function to perform bitonic sort with CUDA
void bitonicSortCUDA(int *h_array, int N) {
    // Allocate memory on the device
    int *d_array;
    hipMalloc(&d_array, N * sizeof(int));

    // Copy array data to the device
    hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice);

    int numThreads = N / 2;  // Each thread handles two elements
    int numBlocks = (N + numThreads - 1) / numThreads;  // Number of blocks to launch

    // Perform the bitonic sort iteratively
    int step_count = log2(N);  // Number of steps for bitonic sort

    // First phase: Each thread processes pairs of elements using the split function
    for (int step = 1; step <= step_count; step++) {
        split(h_array, 0, N, 1);  // Ascending order split
        hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice);
        
        // Perform bitonic merge for this step
        bitonic_merge<<<numBlocks, numThreads>>>(d_array, 0, N, 1);  // Ascending order
        hipDeviceSynchronize();  // Ensure all threads finish before moving on
    }

    // Final pass: Perform bitonic merge iteratively for the entire array size
    int currentSize = 2;
    while (currentSize <= N) {
        bitonic_merge<<<numBlocks, numThreads>>>(d_array, 0, currentSize, 1);  // Ascending order merge
        currentSize *= 2;  // Double the size of the merged section
        hipDeviceSynchronize(); // Ensure proper synchronization
    }

    // Copy the sorted array back to the host
    hipMemcpy(h_array, d_array, N * sizeof(int), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_array);
}

int main(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: %s <log2(size)>\n", argv[0]);
        return -1;
    }

    int q = atoi(argv[1]);
    const int N = 1 << q;

    // Generate a random array
    int *h_array = (int *)malloc(N * sizeof(int));
    generate_random_array(h_array, N);

    // Perform the bitonic sort using CUDA
    bitonicSortCUDA(h_array, N);

    // Print the sorted array
    printf("Sorted Array: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_array[i]);
    }
    printf("\n");

    // Free the allocated memory
    free(h_array);

    return 0;
}
